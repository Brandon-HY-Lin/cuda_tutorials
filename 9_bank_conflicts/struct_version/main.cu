#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>


struct Pehw
{
	float x, y, z, w;
  float padding;
};

__global__
void MyKernel (unsigned long long *time)
{
	__shared__ Pehw shared[1024];
	unsigned long long startTime = std::clock();

	shared[threadIdx.x].x++;

	unsigned long long finishTime = std::clock();

	*time = (finishTime - startTime);
}


int main ()
{
	unsigned long long time;
	unsigned long long *d_time;

	hipMalloc(&d_time, sizeof(unsigned long long));

	for (int i = 0; i < 10; ++i) {
		MyKernel<<<1, 32>>>(d_time);

		hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);

		// 14 is the overhead for calling clock
		std::cout << "Time: " << (time - 14) / 32 << std::endl;

		std::cout << std::endl;
	}

	hipFree(d_time);

	std::cin.get();

	hipDeviceReset();

	return 0;
}
